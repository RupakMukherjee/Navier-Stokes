#include <hipfft/hipfft.h>
#include "cufftUtils.h"
 
// Declared extern "C" to disable C++ name mangling

extern "C" void createCUFFTPlan2D(void *plan, int nx, int ny, int planType, void *stream)
{
    hipfftHandle *cPlan;

    cPlan = (hipfftHandle *)plan;
    CHECK_CUFFT(hipfftPlan2d(cPlan, nx, ny, (hipfftType)planType));
    CHECK_CUFFT(hipfftSetStream(*cPlan,  (hipStream_t)stream));
}

extern "C" void executeCUFFT2D(void *plan, void *iData, void *oData, int planType)
{
    hipfftHandle *cPlan;

    cPlan = ((hipfftHandle *)plan);
    switch (planType)
    {
        case HIPFFT_D2Z: CHECK_CUFFT(hipfftExecD2Z(*cPlan, (hipfftDoubleReal *)iData, (hipfftDoubleComplex *)oData));
                        break;
        case HIPFFT_Z2D: CHECK_CUFFT(hipfftExecZ2D(*cPlan, (hipfftDoubleComplex *)iData, (hipfftDoubleReal *)oData));
                        break;
    }
}

extern "C" void destroyCUFFTPlan2D(void *plan)
{
    hipfftHandle *cPlan;

    cPlan = (hipfftHandle *)plan;
    CHECK_CUFFT(hipfftDestroy(*cPlan));
}  
